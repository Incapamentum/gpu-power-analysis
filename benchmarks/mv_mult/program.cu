#include "../definitions.cuh"
#include "mv_mul.cuh"

void matrixVectorMultiply(int *a, int *b, int *c, int n)
{
    int *a_dev, *b_dev, *c_dev;
    int size;

    size = N_WIDTH * sizeof(int)   ;

    hipMalloc((void **)&a_dev, size * size);
    hipMalloc((void **)&b_dev, size);
    hipMalloc((void **)&c_dev, size);

    hipMemcpy(a_dev, a, size * size, hipMemcpyHostToDevice);
    hipMemcpy(b_dev, b, size, hipMemcpyHostToDevice);

    mv_mul<<<1, 1>>>(a_dev, b_dev, c_dev, n);

    hipFree(a_dev); hipFree(b_dev); hipFree(c_dev);
}

int main(void)
{
    int i, j;
    int *a, *b, *c;

    a = new int[N_WIDTH * N_WIDTH];
    b = new int[N_WIDTH];
    c = new int[N_WIDTH];

    for (i = 0; i < N_WIDTH; i++)
    {
        for (j = 0; j < N_WIDTH; j++)
        {
            a[i * N_WIDTH + j] = i * N_WIDTH + j + 1;
        }

        b[i] = i + 1;
    }

    matrixVectorMultiply(a, b, c, N_WIDTH);

    return 0;
}