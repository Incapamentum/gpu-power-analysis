#include "hip/hip_runtime.h"
#include "kernels.cuh"

__global__
void daxpy(int n, double a, double *x, double *y)
{
    int i;

    i = blockIdx.x * blockDim.x + threadIdx.x;
    if (i < n) y[i] = a * x[i] + y[i];
}