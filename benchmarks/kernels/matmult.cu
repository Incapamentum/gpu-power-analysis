#include "hip/hip_runtime.h"
#include "kernels.cuh"

__global__
void matmult(float *c, float *a, float *b, int n)
{
    int row, col, k;
    float sum = 0.0f;

    row = blockIdx.y * blockDim.y + threadIdx.y;
    col = blockIdx.x * blockDim.x + threadIdx.x;

    for (k = 0; k < n; k++)
        sum += a[row * n + k] * b[k * n + col];

    c[row * n + col] = sum;
}