#include "../definitions.cuh"
#include "matmul.cuh"

void matrixMultiplication(float *c, float *a, float *b, int n)
{
    int size;
    float *cd, *ad, *bd;

    size = n * n * sizeof(float);

    hipMalloc((void **)&ad, size);
    hipMemcpy(ad, a, size, hipMemcpyHostToDevice);
    hipMalloc((void **)&bd, size);
    hipMemcpy(bd, b, size, hipMemcpyHostToDevice);

    hipMalloc((void **)*&cd, size);

    matmult<<<1, 1>>>(cd, ad, bd, n);

    hipFree(cd);
    hipFree(ad);
    hipFree(bd);    
}

int main(void)
{
    float a[WIDTH * WIDTH], b[WIDTH * WIDTH], c[WIDTH * WIDTH];
    int i;

    for (i = 0; i < (WIDTH * WIDTH); i++)
    {
        a[i] = 5;
        b[i] = 5;
        c[i] = 0;
    }

    matrixMultiplication(c, a, b, WIDTH);

    return 0;
}