#include "host.h"

#include "../kernels/kernels.cuh"

void matrixMultiplication(float *c, float *a, float *b, int n)
{
    int size;
    float *cd, *ad, *bd;

    size = n * n * sizeof(float);

    hipMalloc((void **)&ad, size);
    hipMemcpy(ad, a, size, hipMemcpyHostToDevice);
    hipMalloc((void **)&bd, size);
    hipMemcpy(bd, b, size, hipMemcpyHostToDevice);

    hipMalloc((void **)*&cd, size);

    matmult<<<1, 1>>>(cd, ad, bd, n);

    hipFree(cd);
    hipFree(ad);
    hipFree(bd);    
}