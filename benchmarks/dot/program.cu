#include <time.h>

#include "../definitions.cuh"
#include "dot.cuh"

void dotProduct(int *a, int *b, int *c, int size)
{
    int *dev_a, *dev_b, *dev_c;

    hipMalloc((void **)&dev_a, size);
    hipMalloc((void **)&dev_b, size);
    hipMalloc((void **)&dev_c, sizeof(int));

    hipMemcpy(dev_a, a, size, hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, size, hipMemcpyHostToDevice);

    dot<<<1, 1>>>(dev_a, dev_b, dev_c);

    hipFree(dev_a); hipFree(dev_b); hipFree(dev_c);
}

int main(void)
{
    int i;
    int *a, *b, *c;
    int size;

    size = DOT_ELEMENTS * sizeof(int);

    a = (int *)malloc(size);
    b = (int *)malloc(size);
    c = (int *)malloc(sizeof(int));

    srand(time(0));

    for (i = 0; i < DOT_ELEMENTS; i++)
    {
        a[i] = rand() % 100;
        b[i] = rand() % 100;
    }

    dotProduct(a, b, c, size);

    free(a); free(b); free(c);

    return 0;
}