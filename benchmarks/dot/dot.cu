#include "hip/hip_runtime.h"
#include "../definitions.cuh"
#include "kernel.cuh"

__global__
void dot(int *a, int *b, int *c)
{
    int i, sum;
    __shared__ int temp[M];

    temp[threadIdx.x] = a[threadIdx.x] * b[threadIdx.x];

    __syncthreads();

    if (0 == threadIdx.x)
    {
        sum = 0;

        for (i = 0; i < M; i++)
            sum += temp[i];

        *c = sum;

    }
}