#include "hip/hip_runtime.h"
#include "../definitions.cuh"
#include "dot.cuh"

__global__
void dot(int *a, int *b, int *c)
{
    int i, sum;
    __shared__ int temp[M_WIDTH];

    temp[threadIdx.x] = a[threadIdx.x] * b[threadIdx.x];

    __syncthreads();

    if (0 == threadIdx.x)
    {
        sum = 0;

        for (i = 0; i < M_WIDTH; i++)
            sum += temp[i];

        *c = sum;

    }
}