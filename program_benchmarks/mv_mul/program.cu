#include "../definitions.cuh"
#include "mv_mul.cuh"

void matrixVectorMultiply(float *c, float *a, float *b, int n)
{
    float *a_dev, *b_dev, *c_dev;
    int size;

    size = n * sizeof(int);

    hipMalloc((void **)&a_dev, size * size);
    hipMalloc((void **)&b_dev, size);
    hipMalloc((void **)&c_dev, size);

    hipMemcpy(a_dev, a, size * size, hipMemcpyHostToDevice);
    hipMemcpy(b_dev, b, size, hipMemcpyHostToDevice);

    mv_mul<<<1, 1>>>(a_dev, b_dev, c_dev, n);

    hipFree(a_dev); hipFree(b_dev); hipFree(c_dev);
}

int main(void)
{
    int i, j;
    float *a, *b, *c;

    a = new float[N_WIDTH * N_WIDTH];
    b = new float[N_WIDTH];
    c = new float[N_WIDTH];

    for (i = 0; i < N_WIDTH; i++)
    {
        for (j = 0; j < N_WIDTH; j++)
        {
            a[i * N_WIDTH + j] = i * N_WIDTH + j + 1;
        }

        b[i] = i + 1;
    }

    matrixVectorMultiply(c, a, b, N_WIDTH);

    return 0;
}