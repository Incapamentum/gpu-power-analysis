#include "hip/hip_runtime.h"
#include "mv_mul.cuh"

__global__
void mv_mul(int *a, int *b, int *c, int n)
{
    int j, row, sum = 0;

    row = threadIdx.x + blockDim.x * blockIdx.x;

    if (row < n)
        for (j = 0; j < n; j++)
            sum = sum + a[row * n + j] * b[j];

    c[row] = sum;

}