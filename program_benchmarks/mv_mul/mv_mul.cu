#include "hip/hip_runtime.h"
#include "mv_mul.cuh"

__global__
void mv_mul(float *c, float *a, float *b, int n)
{
    int j, row, sum = 0;

    row = threadIdx.x + blockDim.x * blockIdx.x;

    if (row < n)
        for (j = 0; j < n; j++)
            sum = sum + a[row * n + j] * b[j];

    c[row] = sum;
}