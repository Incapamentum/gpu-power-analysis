#include "../definitions.cuh"
#include "saxpy.cuh"

void singlePrecision(int n, float a, float *x, float *y)
{
    float *d_x, *d_y;

    hipMalloc(&d_x, n * sizeof(float));
    hipMemcpy(d_x, x, n * sizeof(float), hipMemcpyHostToDevice);
    hipMalloc(&d_y, n * sizeof(float));
    hipMemcpy(d_y, y, n * sizeof(float), hipMemcpyHostToDevice);

    saxpy<<<1, 1>>>(n, a, d_x, d_y);

    hipFree(d_x);
    hipFree(d_y);
}

int main(void)
{
    float *x, *y;
    int i;

    x = (float *)malloc(SAXPY_ELEMENTS * sizeof(float));
    y = (float *)malloc(SAXPY_ELEMENTS * sizeof(float));

    for (i = 0; i < SAXPY_ELEMENTS; i++)
    {
        x[i] = 1.0f;
        y[i] = 2.0f;
    }

    singlePrecision(SAXPY_ELEMENTS, 2.0f, x, y);

    return 0;    
}

