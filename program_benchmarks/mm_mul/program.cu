#include "../definitions.cuh"
#include "mm_mul.cuh"

void matrixMultiplication(float *c, float *a, float *b, int n)
{
    int size;
    float *cd, *ad, *bd;

    size = n * n * sizeof(float);

    hipMalloc((void **)&ad, size);
    hipMemcpy(ad, a, size, hipMemcpyHostToDevice);
    hipMalloc((void **)&bd, size);
    hipMemcpy(bd, b, size, hipMemcpyHostToDevice);

    hipMalloc((void **)&cd, size);

    mm_mul<<<1, 1>>>(cd, ad, bd, n);

    hipFree(cd);
    hipFree(ad);
    hipFree(bd);    
}

int main(void)
{
    float a[M_WIDTH * M_WIDTH], b[M_WIDTH * M_WIDTH], c[M_WIDTH * M_WIDTH];
    int i;

    for (i = 0; i < (M_WIDTH * M_WIDTH); i++)
    {
        a[i] = 5;
        b[i] = 5;
        c[i] = 0;
    }

    matrixMultiplication(c, a, b, M_WIDTH);

    return 0;
}