#include "../program_benchmarks/definitions.cuh"

#include "../program_benchmarks/mm_mul/mm_mul.cuh"
#include "../program_benchmarks/mv_mul/mv_mul.cuh"

#include <stdio.h>
#include <string.h>

void matrixMultiplication(float *c, float *a, float *b, int n)
{
    float *c_dev, *a_dev, *b_dev;
    int size;

    size = n * n * sizeof(float);

    hipMalloc((void **)&a_dev, size);
    hipMemcpy(a_dev, a, size, hipMemcpyHostToDevice);
    hipMalloc((void **)&b_dev, size);
    hipMemcpy(b_dev, b, size, hipMemcpyHostToDevice);

    hipMalloc((void **)&c_dev, size);

    mm_mul<<<1, 1>>>(c_dev, a_dev, b_dev, n);

    hipFree(c_dev);
    hipFree(a_dev);
    hipFree(b_dev);    
}

void matrixVectorMultiply(float *c, float *a, float *b, int n)
{
    float *a_dev, *b_dev, *c_dev;
    int size;

    size = n * sizeof(int);

    hipMalloc((void **)&a_dev, size * size);
    hipMalloc((void **)&b_dev, size);
    hipMalloc((void **)&c_dev, size);

    hipMemcpy(a_dev, a, size * size, hipMemcpyHostToDevice);
    hipMemcpy(b_dev, b, size, hipMemcpyHostToDevice);

    mv_mul<<<1, 1>>>(a_dev, b_dev, c_dev, n);

    hipFree(a_dev); hipFree(b_dev); hipFree(c_dev);
}

int main(int argc, char **argv)
{
    dim3 Db = dim3(1);
    dim3 Dg = dim3(1);

    int i, j, n = 10;

    // Host variables
    float *a, *b, *c;

    if (argc != 2)
    {
        printf("Insufficient arguments!\n");
        return -1;
    }

    a = (float *)malloc(n * n * sizeof(float));

    // Allocating memory for matrix-matrix
    if (strcmp("mm_mul", argv[1]) == 0)
    {
        b = (float *)malloc(n * n * sizeof(float));
        c = (float *)malloc(n * n * sizeof(float));

        for (i = 0; i < (n * n); i++)
        {
            a[i] = 5;
            b[i] = 5;
            c[i] = 0;
        }

        matrixMultiplication(c, a, b, n);
    }
    else if (strcmp("mv_mul", argv[1]) == 0)
    {
        b = (float *)malloc(n * sizeof(float));
        c = (float *)malloc(n * sizeof(float));

        for (i = 0; i < n; i++)
        {
            for (j = 0; j < n; j++)
            {
                a[i * n + j] = i * n + j + 1;
            }

            b[i] = i + 1;
        }

        matrixVectorMultiply(c, a, b, n);
    }

    // Freeing allocated memory
    free(a); free(b); free(c);

    return 0;
}