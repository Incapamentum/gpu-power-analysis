#include "../program_benchmarks/definitions.cuh"
#include "../program_benchmarks/mm_mul/mm_mul.cuh"

#include "nvmlPower.hpp"

int main(void)
{
    dim3 Db = dim3(1);
    dim3 Dg = dim3(1);

    int n = 10;

    // Host variables
    float *a, *b, *c;

    // Device variables
    float *d_a, *d_b, *d_c;

    // Allocate host memory
    a = (float *)malloc(n * sizeof(float));
    b = (float *)malloc(n * sizeof(float));
    c = (float *)malloc(n * sizeof(float));

    // Allocate device memory
    hipMalloc((void **)&d_a, n * sizeof(float));
    hipMalloc((void **)&d_b, n * sizeof(float));
    hipMalloc((void **)&d_c, n * sizeof(float));

    nvmlAPIRun();
    mm_mul<<<Db, Dg>>>(d_c, d_a, d_b, n);
    nvmlAPIEnd();

    // Freeing allocated memory
    free(a); free(b); free(c);
    hipFree(d_a); hipFree(d_b); hipFree(d_c);

    return 0;
}